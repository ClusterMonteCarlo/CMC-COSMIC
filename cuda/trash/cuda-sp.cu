#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cutil.h>

#define _CUDA_MAIN_

#include "hip/hip_runtime.h"
#include "../cmc.h"
#include "../cmc_vars.h"

__device__ float *cu_m;
__device__ float *cu_m_;

__device__ float *cu_r;
__device__ float *cu_r_;

__device__ float *cu_phi;
__device__ float *cu_phi_;

__device__ float *cu_E;
__device__ float *cu_E_;

__device__ float *cu_J;
__device__ float *cu_J_;

__device__ long  *cu_kmin;
__device__ long  *cu_kmax;
__device__ long  *cu_ktemp;

long totalStars;

__device__ int sign(float a)
{
	return (int)(fabsf(a)/a);
}

__device__ int sign(float a, float aa)
{
	if (fabsf(a) > 0.0000001)
		return sign(a);
	return sign(aa);
}


//-------------------------------------------------------
// Found in CUDA examples / yanked from dsfun90 directly
//-------------------------------------------------------
__device__ inline void D2F_subtract(float &p0, float &p1, float a0, 
	float a1, float b0, float b1)
{
	float t1 = a0 - b0;
	float e = t1 - a0;
	float t2 = ((-b0 - e) + (a0 - (t1 - e))) + a1 - b1;

	p0 = e = t1 + t2;
	p1 = t2 - (e - t1);
}

__device__ inline void D2F_multiply(float &p0, float &p1,
	float a0, float a1, float b0, float b1)
{
	float cona = a0 * 8193.0f;
	float conb = b0 * 8193.0f;
	float sa1 = cona - (cona - a0);
	float sb1 = conb - (conb - b0);
	float sa2 = a0 - sa1;
	float sb2 = b0 - sb1;

	float c11 = a0 * b0;
	float c21 = (((sa1 * sb1 - c11) + sa1 * sb2) + sa2 * sb1) + sa2 * sb2;

	float c2 = a0 * b1 + a1 * b0;
	
	float t1 = c11 + c2;
	float e = t1 - c11;
	float t2 = ((c2 - e) + (c11 - (t1 - e))) + c21 + a1 * b1;

	p0 = e = t1 + t2;
	p1 = t2 - (e - t1);
}

__device__ inline void D2F_divide(float &p, float &pp, 
	float a, float aa, float b, float bb)
{
	float CONST = 4097.0f; //could be one of (2049, 4097, 8193)
	float s1 = a / b;
	float cona = CONST * s1;
	float conb = CONST * b;
	float a1 = cona - (cona - s1);
	float b1 = conb - (conb - b);
	float a2 = s1 - a1;
	float b2 = b - b1;

	float c11 = s1 * b;
	float c21 = (((a1*b1 - c11) + a1*b2) + a2*b1) + a2*b2;
	float c2 = s1 * bb;

	float t1 = c11 + c2;
	float e = t1 - c11;
	float t2 = ((c2-e)+ (c11-(t1-e))) + c21;

	float t12 = t1 + t2;
	float t22 = t2 - (t12 - t1);

	float t11 = a - t12;
	e = t11 - a;
	float t21 = ((-t12-e)+(a-(t11-e))) + aa - t22;
	float s2 = (t11 + t21) / b;
	p = s1 + s2;
	pp = s2 - (p - s1);
}

__device__ void cuPHI_S(float &a, float &aa, float rad, float rad_,
	long nstar, float cu_m, float cu_m_, float cu_r, float cu_r_)
{	
	float n = (float)nstar/2;
	float nn = 0.0f;//(float)(nstar - n);

	float a0, a1, d0, d1;
	D2F_multiply(d0, d1, cu_r, cu_r_, n, nn);
	D2F_divide(a0, a1, cu_m, cu_m_, d0, d1);

	float b0, b1;
	D2F_multiply(d0, d1, rad, rad_, n, nn);
	D2F_divide(b0, b1, cu_m, cu_m_, d0, d1);

	if (rad>=cu_r){	a = b0; aa = b1; }
	else { a = a0; aa = a1; }
}

__device__ float cuPHI_S(float rad, float rad_,
	long nstar, float cu_m, float cu_m_, float cu_r, float cu_r_)
{	
	float n = (float)nstar;
	float nn = (float)(nstar - n);

	float a0, a1, d0, d1;
	D2F_multiply(d0, d1, cu_r, cu_r_, n, nn);
	D2F_divide(a0, a1, cu_m, cu_m_, d0, d1);

	float b0, b1;
	D2F_multiply(d0, d1, rad, rad_, n, nn);
	D2F_divide(b0, b1, cu_m, cu_m_, d0, d1);

	if (rad>=cu_r)
		return b0;
	return a0;
}

__device__ float cuFunction_Q(long j, long k, float E, float E_, float J, float J_,
	long nstar, float *cu_m, float *cu_m_, float *cu_r, float *cu_r_, float *cu_phi, float *cu_phi_)
{
	float a = cu_r_[k];
	float b = cu_r[k];
	float e = a/b;
	float d = (1 - e + e*e - e*e*e)/b;
	return 2.0*(E-(cu_phi[k]+cuPHI_S(cu_r[k], cu_r_[k], nstar, 
			cu_m[j], cu_m_[j], cu_r[j], cu_r_[j])))-((J*d)*(J*d));
}

__device__ void cuFUNC(float &a0, float &a1, long j, long k, float E, float E_, float J, float J_,
	long nstar, float *cu_m, float *cu_m_, float *cu_r, float *cu_r_, float *cu_phi, float *cu_phi_)
{
	float a, aa, b, bb, l, ll, m, mm, f, ff, r, rr, p, pp, r2, rr2;

	D2F_subtract(a, aa, E, E_, cu_phi[k], cu_phi_[k]);
	cuPHI_S(p, pp, cu_r[k], cu_r_[k], nstar, cu_m[j], cu_m_[j], cu_r[j], cu_r_[j]);

	D2F_subtract(b, bb, a, aa, p, pp);
	D2F_multiply(l, ll, J, J_, J, J_);
	//D2F_multiply(r, rr, cu_r[k], cu_r_[k], cu_r[k], cu_r_[k]);
	r = cu_r[k]*cu_r[k]; rr =0.0f;
	D2F_multiply(r2, rr2, r, rr, 2.0, 0.0);
	D2F_multiply(m, mm, b, bb, r2, rr2);
	D2F_subtract(f, ff, m, mm, l, ll);

	a0 = f;
	a1 = ff;
}

__device__ float cuFUNC(long j, long k, float E, float E_, float J, float J_,
	long nstar, float *cu_m, float *cu_m_, float *cu_r, float *cu_r_, float *cu_phi, float *cu_phi_)
{
	float a, aa, b, bb, l, ll, m, mm, f, ff, r, rr, p, pp, r2, rr2;

	D2F_subtract(a, aa, E, E_, cu_phi[k], cu_phi_[k]);
	cuPHI_S(p, pp, cu_r[k], cu_r_[k], nstar, cu_m[j], cu_m_[j], cu_r[j], cu_r_[j]);

	D2F_subtract(b, bb, a, aa, p, pp);
	D2F_multiply(l, ll, J, J_, J, J_);
	//D2F_multiply(r, rr, cu_r[k], cu_r_[k], cu_r[k], cu_r_[k]);
	r = cu_r[k]*cu_r[k]; rr =0.0f;
	D2F_multiply(r2, rr2, r, rr, 2.0, 0.0);
	D2F_multiply(m, mm, b, bb, r2, rr2);
	D2F_subtract(f, ff, m, mm, l, ll);

	return f;
}

//========================================================================
//start up CUDA - needs to be called before anything else
//========================================================================
void hipInit()
{
	printf("------------------ USING CUDA -------------------\n");
	printf("\tInitializing devices...\n");
	CUT_DEVICE_INIT();

	dfile  = fopen("Qdiff.csv", "w+");
	dfile2 = fopen("kdiff.csv", "w+");
	fprintf(dfile, "Star Index, Device kmin, Host kmin, Device kmax, Host kmax\n");
	fprintf(dfile2, "Star Index, Host kmin, Device kmin, Host kmax, Device kmax\n");

	printf("\tCreating timer...\n");
	cutCreateTimer(&timer);
	totalStars = clus.N_STAR*2; //there can be no more than twice the number of initial stars... i hope

	m = (float*)malloc(sizeof(float)*totalStars);
	m_ = (float*)malloc(sizeof(float)*totalStars);
	r = (float*)malloc(sizeof(float)*totalStars);
	r_ = (float*)malloc(sizeof(float)*totalStars);
    phi = (float*)malloc(sizeof(float)*totalStars);
    phi_ = (float*)malloc(sizeof(float)*totalStars);
	cE = (float*)malloc(sizeof(float)*totalStars);
	cE_ = (float*)malloc(sizeof(float)*totalStars);
	cJ = (float*)malloc(sizeof(float)*totalStars);
	cJ_ = (float*)malloc(sizeof(float)*totalStars);

	h_kmin = (long*)malloc(sizeof(long)*totalStars);
	h_kmax = (long*)malloc(sizeof(long)*totalStars);
	h_ktemp = (long*)malloc(sizeof(long)*totalStars);

	printf("\tInitializing device memory for 2*%i objects (%ikB)...\n", totalStars/2, totalStars*15/8000);
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_m, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_m_, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_r, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_r_, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_phi, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_phi_, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_E, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_E_, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_J, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_J_, sizeof(float)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_kmin, sizeof(long)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_kmax, sizeof(long)*totalStars) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &cu_ktemp, sizeof(long)*totalStars) );
	printf("-------------------------------------------------\n");
}

//========================================================================
//copies all of the star info to the device at once
//hopefully this isn't a very big factor, who knows
//========================================================================
void cuCopyDataToDevice()
{
	for (long si = 1; si <= clus.N_MAX_NEW; si++)
	{
		float E = star[si].E + PHI_S(star[si].r, si);
		cE[si] = (float)(E);
		cE_[si] = (float)(E - cE[si]);

		cJ[si] = (float)star[si].J;
		cJ_[si] = (float)(star[si].J - cJ[si]);

		m[si] = (float)star[si].m;
		m_[si] = (float)(star[si].m - m[si]);

		r[si] = (float)star[si].r;
		r_[si] = (float)(star[si].r - r[si]);

		phi[si] = (float)star[si].phi;
		phi_[si] = (float)(star[si].phi - phi[si]);
	}

	CUDA_SAFE_CALL( hipMemcpy( cu_m, m, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( cu_m_, m_, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( cu_r, r, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( cu_r_, r, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( cu_phi, phi, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( cu_phi_, phi_, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( cu_E, cE, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( cu_E_, cE_, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( cu_J, cJ, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( cu_J_, cJ_, sizeof(float)*totalStars, hipMemcpyHostToDevice) );
}

//========================================================================
//copies all of the final products back for processing
//========================================================================
void cuCopyDataToHost()
{
    CUDA_SAFE_CALL( hipMemcpy( h_kmin, cu_kmin, sizeof(long)*totalStars, hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy( h_kmax, cu_kmax, sizeof(long)*totalStars, hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy( h_ktemp, cu_ktemp, sizeof(long)*totalStars, hipMemcpyDeviceToHost) );
}


//TODO - rewrite linear search aspect of calc_orbit_rs and find all ktemps before hand
//if Qtemp is still negative after linear search??? then use negative ktemps and test for that
//later
//========================================================================
// The equivalent of FindZero_Q
//========================================================================
__global__ void cuFindZero_Q(long start, long nstar, float *cu_m, float *cu_m_, 
		float *cu_r, float *cu_r_, float *cu_phi, float *cu_phi_, float *cu_E, 
		float *cu_E_, float *cu_J, float *cu_J_, long *cu_kmin, long *cu_kmax, long *cu_ktemp)
{

	long n = nstar;
	long si = threadIdx.x + blockDim.x * blockIdx.x + start;
	long j = si;

	float E, J, t, tt, E_, J_; 
	long ktry, kmin, kmax, sa, sb, k1, k2;
	E = cu_E[j];  	E_ = cu_E_[j];
	J = cu_J[j];	J_ = cu_J_[j];

/*
//=============================================================
	if(cuFUNC(j, kmin, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_)<
		cuFUNC(j, kmax, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_)){
		do {
			ktry = (kmin+kmax+1)/2;
			if (cuFUNC(j, ktry, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_)<0){
				kmin = ktry;
			} else {
				kmax = ktry-1;
			}
		} while (kmax!=kmin);
	} else {
		do {
			ktry = (kmin+kmax+1)/2;
			if (cuFUNC(j, ktry, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_)>0){
				kmin = ktry;
			} else {
				kmax = ktry-1;
			}
		} while (kmax!=kmin);
	}

	k1 = kmin;

	kmin = cu_ktemp[si];
	kmax = n;

	if(cuFUNC(j, kmin, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_)<
		cuFUNC(j, kmax, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_)){
		do {
			ktry = (kmin+kmax+1)/2;
			if (cuFUNC(j, ktry, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_)<0){
				kmin = ktry;
			} else {
				kmax = ktry-1;
			}
		} while (kmax!=kmin);
	} else {
		do {
			ktry = (kmin+kmax+1)/2;
			if (cuFUNC(j, ktry, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_)>0){
				kmin = ktry;
			} else {
				kmax = ktry-1;
			}
		} while (kmax!=kmin);
	}

	k2 = kmin;
//============================================================
*/
	
	//---------------------------------------------
	// search for kmin first 
	//---------------------------------------------
	kmin = 0;
	kmax = cu_ktemp[si];	

	cuFUNC(t, tt, j, kmin, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_);
	sa = sign(t, tt);
	cuFUNC(t, tt, j, kmax, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_);
	sb = sign(t, tt);

	//20 loops should be enough for at least 
	for (int i=0; i<=MAX_LOOPS; i++)
	{	
		ktry = (kmin+kmax+1)/2;
		cuFUNC(t, tt, j, ktry, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_);
		kmin = (1 + sa*sign(t, tt))/2 * (ktry - kmin) + kmin;
		kmax = (1 + sb*sign(t, tt))/2 * (ktry - kmax - 1) + kmax;
	}
	k1 = kmax;

	//---------------------------------------------
	// search for kmax next 
	//---------------------------------------------
	kmin = cu_ktemp[si];
	kmax = n;
	
	cuFUNC(t, tt, j, kmin, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_);
	sa = sign(t, tt);
	cuFUNC(t, tt, j, kmax, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_);
	sb = sign(t, tt);

	//24 loops should be enough for at least 10^7 stars
	for (int i=0; i<=MAX_LOOPS; i++)
	{	
		ktry = (kmin+kmax+1)/2;
		cuFUNC(t, tt, j, ktry, E, E_, J, J_, n, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_);
		kmin = (1 + sa*sign(t, tt))/2 * (ktry - kmin) + kmin;
		kmax = (1 + sb*sign(t, tt))/2 * (ktry - kmax - 1) + kmax;
	}
	k2 = kmin;

	cu_kmin[j] = k1;
	cu_kmax[j] = k2;
}


//========================================================================
// prepares the ktemps for later
//========================================================================
__global__ void cuFindKtemps(long start, long nstar, float *cu_m, float *cu_m_, 
		float *cu_r, float *cu_r_, float *cu_phi, float *cu_phi_, float *cu_E, 
		float *cu_E_, float *cu_J, float *cu_J_, long *cu_ktemp)
{
	long si = threadIdx.x + blockDim.x * blockIdx.x + start;

//	float E, J, Qtemp, E_, J_;
	long ktemp = si;
//	E = cu_E[si];  	E_ = cu_E_[si];
//	J = cu_J[si];	J_ = cu_J_[si];
   	
/*	Qtemp = cuFunction_Q(si, ktemp, E, E_, J, J_, nstar, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_);
	if (Qtemp < 0.0) 
	{
		ktemp = -1;
		do {
			ktemp++;
			Qtemp = cuFunction_Q(si, ktemp, E, E_, J, J_, nstar, cu_m, cu_m_, cu_r, cu_r_, cu_phi, cu_phi_);
		} while (Qtemp < 0.0 && ktemp <= nstar);		
		if (ktemp >= nstar) 
			ktemp = si;
	}*/

	cu_ktemp[si] = ktemp;
}

//========================================================================
// Host frontend for finding the k-values
//========================================================================
void cuCalculateKs()
{
	printf("------------------ USING CUDA -------------------\n");

	cutResetTimer(timer);
	cutStartTimer(timer);
	cuCopyDataToDevice();

	for (long start = 1; start <= clus.N_MAX_NEW; start += THREADS)
		cuFindKtemps<<< GRID_DIM, BLOCK_DIM >>>(start, clus.N_MAX, cu_m, cu_m_, cu_r, 
				cu_r_, cu_phi, cu_phi_, cu_E, cu_E_, cu_J, cu_J_, cu_ktemp);

	hipDeviceSynchronize();

	for (long start = 1; start <= clus.N_MAX_NEW; start += THREADS)
		cuFindZero_Q<<< GRID_DIM, BLOCK_DIM >>>(start, clus.N_MAX, cu_m, cu_m_, cu_r, 
				cu_r_, cu_phi, cu_phi_, cu_E, cu_E_, cu_J, cu_J_, cu_kmin, cu_kmax, cu_ktemp);

	hipDeviceSynchronize();
	cuCopyDataToHost();
	cutStopTimer(timer);
	float time = cutGetTimerValue(timer);
	printf("\tCUDA device time: %f\n", time);
	printf("-------------------------------------------------\n");
}

//========================================================================
// clean up all the CUDA crud
//========================================================================
void cuCleanUp(int argc, char **argv)
{
	printf("------------------ USING CUDA -------------------\n");
	printf("\tFreeing device memory...\n");

    free(r);
	free(r_);
    free(m);
    free(phi);
	free(cE);
	free(cJ);
	free(h_kmin);
	free(h_kmax);
	free(h_ktemp);

    CUDA_SAFE_CALL( hipFree(cu_m) );
    CUDA_SAFE_CALL( hipFree(cu_r) );
	CUDA_SAFE_CALL( hipFree(cu_r_) );
    CUDA_SAFE_CALL( hipFree(cu_phi) );
    CUDA_SAFE_CALL( hipFree(cu_J) );
    CUDA_SAFE_CALL( hipFree(cu_E) );
	CUDA_SAFE_CALL( hipFree(cu_kmin) );
	CUDA_SAFE_CALL( hipFree(cu_kmax) );
	CUDA_SAFE_CALL( hipFree(cu_ktemp) );

	printf("\tReleasing devices...\n");
    CUT_EXIT(0, 0);
	printf("-------------------------------------------------\n");
}

