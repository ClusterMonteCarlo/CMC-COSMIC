#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define _CUDA_MAIN_

#include "cmc_cuda.h" 
#include "../cmc.h"
#include "../cmc_vars.h"

__device__ double *cu_m;
__device__ double *cu_r;
__device__ double *cu_phi;
__device__ double *cu_E;
__device__ double *cu_J;

__device__ long  *cu_kmin;
__device__ long  *cu_kmax;
__device__ long  *cu_ktemp;

long totalStars;

__device__ int sign(double a) {
    return (a < 0 ? -1 : 1);
}


//=========================================================
// Some inline helper functions to keep it clean
//=========================================================
__device__ double cuSQR(double x){
    return x*x;
}

__device__ double cuPHI_S(double rad, long nstar, 
			  double cu_m, double cu_r) //8 flop
{	
    double a = cu_m/nstar;
    if (rad >= cu_r)
	return a/rad;
    return a/cu_r;
}

__device__ double cuFunction_Q(long j, long k, double E, double J, 
			       long nstar, double *cu_m, double *cu_r, double *cu_phi)
{
    double a = cu_r[k];
    return 2.0*(E-(cu_phi[k]+cuPHI_S(a, nstar, cu_m[j],cu_r[j])))-((J/a)*(J/a));
}

#ifndef EXPERIMENTAL
__device__ double cuFUNC(long j, long k, double E, double J, long nstar,  //15 flop
			 double *cu_m, double *cu_r, double *cu_phi)
{
    return 2.0 * cuSQR(cu_r[k]) 
	* (E - cu_phi[k] + cuPHI_S(cu_r[k], nstar, cu_m[j], cu_r[j])) - cuSQR(J);
}
#else
__device__ double cuFUNC(long j, long k, double E, double J, long nstar,  // 21 flop
			 double *cu_m, double *cu_r, double *cu_phi)
{
    return 2.0 * (E - cu_phi[k] + cuPHI_S(cu_r[k], nstar, cu_m[j], cu_r[j])) - cuSQR(J/cu_r[k]);
}
#endif


//========================================================================
// The equivalent of FindZero_Q
//========================================================================
__global__ void cuFindZero_Q(long start, long nstar, double *cu_m, double *cu_r, double *cu_phi, 
			     double *cu_E, double *cu_J, long *cu_kmin, long *cu_kmax, long *cu_ktemp)
{
    
    long n = nstar;
    long si = threadIdx.x + blockDim.x * blockIdx.x + start;
    long j = si;
    
    // should load these values to shared memory? check size of it
    // also, since we dont change E, J can they go into constant memory -> faster?
    // scratch that, no constant memory, maybe texture memory space
    double E, J, t;
    long ktry, kmin, kmax, sa, sb, k1, k2;
    E = cu_E[j];
    J = cu_J[j];
    
    //---------------------------------------------
    // search for kmin first 
    //---------------------------------------------
    kmin = 0;
    kmax = si;//cu_ktemp[si];	
    
    t = cuFUNC(j, kmin, E, J, n, cu_m, cu_r, cu_phi); 
    sa = sign(t);
    t = cuFUNC(j, kmax, E, J, n, cu_m, cu_r, cu_phi);
    sb = sign(t);
    
#pragma unroll 25
    //20 loops should be enough for at least
    for (long i=0; i<=MAX_LOOPS; i++)
	{	
	    ktry = (kmin+kmax+1)/2;
	    t=cuFUNC(j, ktry, E, J, n, cu_m, cu_r, cu_phi);
	    kmin = (1 + sa*sign(t))/2 * (ktry - kmin) + kmin;
	    kmax = (1 + sb*sign(t))/2 * (ktry - kmax - 1) + kmax;
	}
    k1 = kmax;
    
    // ---
    // try #2 on calculations
    // 3 (si) + 9 + 9 (t,sa,sb) + (3+9+13)*30 (loops)
    // 9 + 9 + (3+9+13)*30
    // = 1539 * 5e5 = 0.7695 GLOPs
    // 12.8 GFLOPS
    // ---

    //-------------------------------------------------------------------
    // 44 flop for initial t, sa, sb
    // MAX_LOOPS * (6 + 21 + 19 + 1 (i++)) = 1380 flop + 30flop
    // + 4 extra that were elsewhere
    // 
    // total flops for all of FindZero_Q = (44+1380+30+4)*2 = 2916
    // total flops performed for the search space = 2916 * N = 1.458GFLOP
    // total GFLOPS = 1.454 GFLOP / 0.060sec = 24.6GFLOPS
    //-------------------------------------------------------------------


    //---------------------------------------------
    // search for kmax next 
    //---------------------------------------------
    kmin = si;//cu_ktemp[si];
    kmax = nstar;
    
    t=cuFUNC(j, kmin, E, J, n, cu_m, cu_r, cu_phi);
    sa = sign(t);
    t=cuFUNC(j, kmax, E, J, n, cu_m, cu_r,  cu_phi)-(1e-8);
    sb = sign(t);
    
#pragma unroll 25
    //24 loops should be enough for at least 10^7 stars
    for (long i=0; i<=MAX_LOOPS; i++)
	{	
	    ktry = (kmin+kmax+1)/2;
	    t=cuFUNC(j, ktry, E, J, n, cu_m, cu_r, cu_phi);
	    kmin = (1 + sa*sign(t))/2 * (ktry - kmin) + kmin;
	    kmax = (1 + sb*sign(t))/2 * (ktry - kmax - 1) + kmax;
	}
    k2 = kmax;
    
    cu_kmin[j] = k1;
    cu_kmax[j] = k2;
}


//========================================================================
// prepares the ktemps for later
//========================================================================
__global__ void cuFindKtemps(long start, long nstar, double *cu_m, double *cu_r, 
			     double *cu_phi, double *cu_E, double *cu_J, long *cu_ktemp)
{
    long si = threadIdx.x + blockDim.x * blockIdx.x + start;
    
    //double E, J, Qtemp;
    long ktemp = si;
    //E = cu_E[si];
    //J = cu_J[si];
    
    /*    Qtemp = cuFunction_Q(si, ktemp, E, J, nstar, cu_m, cu_r, cu_phi);
    if (Qtemp < 0.0) 
	{
	    ktemp = -1;
	    do {
		ktemp++;
		Qtemp = cuFunction_Q(si, ktemp, E, J, nstar, cu_m, cu_r, cu_phi);
	    } while (Qtemp < 0.0 && ktemp <= nstar);		
	    if (ktemp >= nstar) 
		ktemp = si;
		}*/
    
    cu_ktemp[si] = ktemp;
}

//========================================================================
// Start up CUDA - needs to be called before anything else
//========================================================================
void cuInitialize()
{
    printf("------------------ USING CUDA -------------------\n");
    printf("\tInitializing devices...\n");
    
    CUT_DEVICE_INIT();
   
    printf("\tFreeing memory for host...\n");
    totalStars = clus.N_STAR*2; //there can be no more than twice the number of initial stars... i hope
    
    m = (double*)malloc(sizeof(double)*totalStars);
    r = (double*)malloc(sizeof(double)*totalStars);
    phi = (double*)malloc(sizeof(double)*totalStars);
    cE = (double*)malloc(sizeof(double)*totalStars);
    cJ = (double*)malloc(sizeof(double)*totalStars);
    
    h_kmin = (long*)malloc(sizeof(long)*totalStars);
    h_kmax = (long*)malloc(sizeof(long)*totalStars);
    h_ktemp = (long*)malloc(sizeof(long)*totalStars);
    
    printf("\tInitializing device memory for 2*%i objects (%ikB)...\n", totalStars/2, totalStars*15/8000);
    CUDA_SAFE_CALL( hipMalloc( (void**) &cu_m, sizeof(double)*totalStars) );
    CUDA_SAFE_CALL( hipMalloc( (void**) &cu_r, sizeof(double)*totalStars) );
    CUDA_SAFE_CALL( hipMalloc( (void**) &cu_phi, sizeof(double)*totalStars) );
    CUDA_SAFE_CALL( hipMalloc( (void**) &cu_E, sizeof(double)*totalStars) );
    CUDA_SAFE_CALL( hipMalloc( (void**) &cu_J, sizeof(double)*totalStars) );
    CUDA_SAFE_CALL( hipMalloc( (void**) &cu_kmin, sizeof(long)*totalStars) );
    CUDA_SAFE_CALL( hipMalloc( (void**) &cu_kmax, sizeof(long)*totalStars) );
    CUDA_SAFE_CALL( hipMalloc( (void**) &cu_ktemp, sizeof(long)*totalStars) );
    hipError_t er = hipGetLastError();
    printf("\tCUDA's comments regarding init: %s\n", hipGetErrorString(er));
    printf("-------------------------------------------------\n");
}

//========================================================================
//copies all of the star info to the device at once
//hopefully this isn't a very big factor, who knows
//========================================================================
void cuCopyDataToDevice()
{
    for (long si = 1; si <= clus.N_MAX_NEW; si++)
	{
	    cE[si] = star[si].E + PHI_S(star[si].r, si);
	    cJ[si] = star[si].J;
	    m[si] = star[si].m;
	    r[si] = star[si].r;
	    phi[si] = star[si].phi;
	}
    
    CUDA_SAFE_CALL( hipMemcpy( cu_m, m, sizeof(double)*totalStars, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( cu_r, r, sizeof(double)*totalStars, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( cu_phi, phi, sizeof(double)*totalStars, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( cu_E, cE, sizeof(double)*totalStars, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( cu_J, cJ, sizeof(double)*totalStars, hipMemcpyHostToDevice) );
}

//========================================================================
//copies all of the final products back for processing
//========================================================================
void cuCopyDataToHost()
{
    CUDA_SAFE_CALL( hipMemcpy( h_kmin, cu_kmin, sizeof(long)*totalStars, hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy( h_kmax, cu_kmax, sizeof(long)*totalStars, hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy( h_ktemp, cu_ktemp, sizeof(long)*totalStars, hipMemcpyDeviceToHost) );
}


//========================================================================
// Host frontend for finding the k-values
//========================================================================
void cuCalculateKs()
{
    printf("------------------ USING CUDA -------------------\n");
    
    //clock_t start = clock();
    cuCopyDataToDevice();
    //clock_t end = clock();
    //printf("\tCUDA memcpy time: %f sec\n", double(end-start)/(CLOCKS_PER_SEC));
    
    for (long start = 1; start <= clus.N_MAX_NEW; start += THREADS)
    	cuFindKtemps<<< GRID_DIM, BLOCK_DIM >>>(start, clus.N_MAX, cu_m, cu_r, 
    						cu_phi, cu_E, cu_J, cu_ktemp);
    hipDeviceSynchronize();
    
    /*    int grids[6] = {8, 16, 20, 24, 28, 32};
    int blocks[7] = {32, 64, 128, 192, 256, 296, 384}; 
    for (int i=0; i<6; i++) {
	for (int j=0; j<7; j++){
	    clock_t begin = clock();
	    for (int x = 0; x<10; x++){
    */
    for (long start = 1; start <= clus.N_MAX_NEW; start += THREADS) //(grids[i]*blocks[j]))
	cuFindZero_Q<<< GRID_DIM, BLOCK_DIM >>>(start, clus.N_MAX+1, cu_m, cu_r, 
						cu_phi, cu_E, cu_J, cu_kmin, 
						cu_kmax, cu_ktemp);
    hipDeviceSynchronize();
		/*		
		
	    }
	    clock_t last = clock();
	    printf("\t%i x %i : %f sec", grids[i], blocks[j], double(last-begin)/(CLOCKS_PER_SEC*10));
	    hipError_t er = hipGetLastError();
	    printf("\t||\tCUDA says: %s\n", hipGetErrorString(er));
	}
	}*/
    
    cuCopyDataToHost();
    hipError_t er = hipGetLastError();
    printf("\tCUDA says: %s\n", hipGetErrorString(er));

    printf("-------------------------------------------------\n");
}

//========================================================================
// clean up all the CUDA crud
//========================================================================
void cuCleanUp()
{
    printf("------------------ USING CUDA -------------------\n");
    printf("\tFreeing device memory...\n");
    
    free(r);
    free(m);
    free(phi);
    free(cE);
    free(cJ);
    free(h_kmin);
    free(h_kmax);
    free(h_ktemp);
    
    CUDA_SAFE_CALL( hipFree(cu_m) );
    CUDA_SAFE_CALL( hipFree(cu_r) );
    CUDA_SAFE_CALL( hipFree(cu_phi) );
    CUDA_SAFE_CALL( hipFree(cu_J) );
    CUDA_SAFE_CALL( hipFree(cu_E) );
    CUDA_SAFE_CALL( hipFree(cu_kmin) );
    CUDA_SAFE_CALL( hipFree(cu_kmax) );
    CUDA_SAFE_CALL( hipFree(cu_ktemp) );
    
    printf("\tReleasing devices...\n");
    printf("-------------------------------------------------\n");
}

